#include "hip/hip_runtime.h"

#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cstdlib>
#include <chrono> 
#include <math.h>
#include <iostream>

using namespace std;
using namespace std::chrono; 

const short maxThreadPerblock=1024;
struct hipGraphicsResource *cuda_vbo_resource;
GLuint points_vbo;
float* d_vbo_ptr = 0;

const unsigned short dim=3; //nbr transformation
const short level=5; //nbr iteration
const unsigned short sizeV=9; //size of polygone 
#define sizeTL (27) //size of transformation

const float h_v[sizeV]={1.0, 0.0, 0.0, 
						0.0, 1.0, 0.0, 
						0.0, 0.0, 1.0
						};
const float h_tl[sizeTL] = {1.0, 0.5, 0.5,
							0.0, 0.5, 0.0,
							0.0, 0.0, 0.5,//T0
    						0.5, 0.0, 0.0,
     						0.5, 1.0, 0.5,
     						0.0, 0.0, 0.5,//T1
   							0.5, 0.0, 0.0,
    						0.0, 0.5, 0.0,
     						0.5, 0.5, 1.0 //T2
							};
const short h_tlSize[dim]={0,9,18};

__constant__ float d_v[sizeV];//device verteses
__constant__ float d_tl[sizeTL];//device transformation list
__constant__ short d_offsetT[dim];
__constant__ short d_sizeV;


__global__ void IFSkernel(float *ver,short level,unsigned short dim, unsigned int Bi,size_t offset);

hipError_t DFS(int thread, unsigned int threadPerblock,unsigned int block,unsigned int Bi,size_t offset,unsigned short mode);

void setUpCamera();

int main(void)
{
    size_t threads=pow(dim,level);
	unsigned int blocks= threads/maxThreadPerblock;
	unsigned short mode=threads%maxThreadPerblock;
	size_t offset=0;
	unsigned int threadPerblock;
	
    if(blocks==0){
        blocks=1;
        threadPerblock=threads;		
    }else{
        threadPerblock=maxThreadPerblock;
        if(mode!=0){
            offset=blocks*maxThreadPerblock;
        }
    }

    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(640, 480, "Hello World", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);

    /* Init glew */
    if(glewInit() != GLEW_OK)
        std::cout << "glewInit error" << std::endl;
    
    setUpCamera();

	// initialize a VBO
	points_vbo = 0;
	// generate 1 VBO buffer
	glGenBuffers(1, &points_vbo); 
	// bind points_vbo to GL_ARRAY_BUFFER. 
	glBindBuffer(GL_ARRAY_BUFFER, points_vbo);
	// locate the memory without initialize the values  
	glBufferData(GL_ARRAY_BUFFER, threads*9 * sizeof(float), 0, GL_DYNAMIC_DRAW);

    
    

    std::cout << "Iteration : " << level << std::endl;
    std::cout << "nbr Transformations : " << dim << std::endl;	
    std::cout << "Total Thread : " << threads << std::endl;
    std::cout << "nbr Block : " << blocks << std::endl;	
    std::cout << "Nbr Thread/Block : " << threadPerblock << std::endl;			
    std::cout << "mode : " << mode << std::endl;
    std::cout << "offset : " << offset << std::endl;	    
    
    auto start = high_resolution_clock::now(); 
    DFS(threads,threadPerblock,blocks,threads/dim,offset,mode);
    auto stop = high_resolution_clock::now(); 
    auto duration = duration_cast<microseconds>(stop - start)*pow(10,-6);
    cout <<"\nExecution time : " << duration.count() << "\n";
    
	glEnableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, points_vbo);
	glVertexAttribPointer(
		0,                  // attribute 0. No particular reason for 0, but must match the layout in the shader.
		3,                  // size
		GL_FLOAT,           // type
		GL_FALSE,           // normalized?
		0,                  // stride
		(void*)0            // array buffer offset
	);

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {        
        glClear(GL_COLOR_BUFFER_BIT);
                        
        // Draw the triangle !
        glDrawArrays(GL_TRIANGLES, 0, threads*3); // Starting from vertex 0; 3 vertices for each triangle              

        /* Swap front and back buffers */
        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();
    }

    glfwTerminate();
    return 0;
}

__global__ void IFSkernel(float *ver,short level,unsigned short dim, unsigned int Bi,size_t offset){	
	size_t N=threadIdx.x + blockIdx.x * blockDim.x + offset;
	size_t n=N;	
	unsigned short T;
	short nbrVertex=d_sizeV/3;
	float *poly=new float[d_sizeV];
	float *p=new float[d_sizeV];

	memcpy(p, d_v, sizeof(float)*d_sizeV);	
	
	while(level>=0){
		T=n/Bi;

		for (short r = 0; r < nbrVertex ; r++)
		{
			for (short c = 0; c < 3 ; c++)
			{							
				poly[r*3+c]=0;	
				for (short k = 0; k < 3 ; k++)
					poly[r*3+c]+=d_tl[d_offsetT[T]+r*3+k] * p[k*3+c];				
			}			
		}

		n=n%Bi;
		Bi=Bi/dim;
		level--;
		memcpy(p, poly, sizeof(float)*d_sizeV);				
	}
	//insert vertices in vbo
	for(short i=0;i<nbrVertex;i++)
	    for(short j=0;j<3;j++) ver[N*d_sizeV+i*3+j]=poly[i+j*3];	 	
}


hipError_t DFS(int threads,unsigned int threadPerblock,unsigned int block,unsigned int Bi,size_t offset,unsigned short mode) {	

	hipError_t cudaStatus;	
	
	hipMemcpyToSymbol(HIP_SYMBOL(d_v), h_v, sizeof(float)*sizeV);
	hipMemcpyToSymbol(HIP_SYMBOL(d_tl), h_tl, sizeof(float)*sizeTL);
	hipMemcpyToSymbol(HIP_SYMBOL(d_offsetT), h_tlSize, sizeof(short)*dim);
	hipMemcpyToSymbol(HIP_SYMBOL(d_sizeV), &sizeV, sizeof(short));

	//connet cuda_vbo_resource to points_vbo
    hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, points_vbo, cudaGraphicsMapFlagsNone);  
	//give access authority of points_vbo to cuda
    hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
    
    size_t num_bytes;  
	//"verteses" points to the GPU memory data store of VBO (points_vbo) maped by cuda_vbo_resource 
    hipGraphicsResourceGetMappedPointer((void **)&d_vbo_ptr, &num_bytes, cuda_vbo_resource); 

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
	    fprintf(stderr, "hipMemcpyToSymbol error: %s\n", hipGetErrorString(cudaStatus));
	    goto Error;
	}

	IFSkernel<<<block,threadPerblock >>> (d_vbo_ptr,level-1,dim,Bi,0);
	
	if(offset!=0){
		IFSkernel<<<1,mode >>> (d_vbo_ptr,level-1,dim,Bi,offset);
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
	    fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	    goto Error;
	}	

 	Error:
	hipFree(d_v);
	hipFree(d_tl);
	hipFree(d_offsetT);

	return cudaStatus;
}

void setUpCamera(){
    glPushMatrix();
    glMatrixMode(GL_PROJECTION);
    gluPerspective(30, 1, 0.1, 500);
    gluLookAt(2, 2, 2, 0, 0.2, 0, 0, 1, 0);
    glPopMatrix();
}