#include <GL/glut.h>
#include <cstdlib>
#include <ctime>
#include <armadillo>
#include <hip/hip_runtime.h>
#include <stack>

//files.associations

using namespace std;

// menu item
#define MENU_SMOOTH 1
#define MENU_FLAT 0

struct fractalLevel
{
    vector<arma::Mat<GLfloat>> tList;
    arma::Mat<GLfloat> trgl;
    int iteration;
};

struct transformLevel
{
    vector<int> trans;
    int iteration;
};

arma::Mat<GLfloat> Triangle = {{0.0, 0.0, 0.0},
                               {1.0, 0.0, 0.0},
                               {0.0, 1.0, 0.0}};

arma::Mat<GLfloat> Triangle2 = {{1.0, 0.0, 0.0},
                                {0.0, 1.0, 0.0},
                                {0.0, 0.0, 1.0}};

vector<arma::Mat<GLfloat>> transfMat{
    {{1.0, 0.5, 0.5},
     {0.0, 0.5, 0.0},
     {0.0, 0.0, 0.5}},
    {{0.5, 0.0, 0.0},
     {0.5, 1.0, 0.5},
     {0.0, 0.0, 0.5}},
    {{0.5, 0.0, 0.0},
     {0.0, 0.5, 0.0},
     {0.5, 0.5, 1.0}}};

double colors[1000] = {0};

int iterations = 0, maxIteration = 11;
double zoom = 1;
int shading = GL_SMOOTH;

// Function prototypes
void generateColors();
double random(bool reset);
void keyboard(unsigned char key, int x, int y);
//void special(int key, int x, int y);
void mouse(int button, int state, int x, int y);
void menu(int item);
void display();
void init();
GLfloat **toGLfloatPoints(arma::Mat<GLfloat> armapoly, int n_row);
void drawPolygone(GLfloat **poly, int n_row);

void divideTriangle(arma::Mat<GLfloat> trgl, vector<arma::Mat<GLfloat>> TransfList, int iteration)
{
    if (iteration == 0)
    {
        trgl = trgl.t();
        int n_rows = trgl.n_rows;
        GLfloat **poly = toGLfloatPoints(trgl, n_rows);
        drawPolygone(poly, n_rows);
    }
    else
    {
        divideTriangle(TransfList[0] * trgl, TransfList, iteration - 1);
        divideTriangle(TransfList[1] * trgl, TransfList, iteration - 1);
        divideTriangle(TransfList[2] * trgl, TransfList, iteration - 1);
    }
}

void divideTriangleIterative(arma::Mat<GLfloat> trgl, vector<arma::Mat<GLfloat>> TransfList, int iter)
{
    stack<fractalLevel> stk;
    fractalLevel level;
    while (true)
    {
        while (iter > 0)
        {
            iter--;
            level.iteration = iter;
            //level.tList = TransfList;
            level.trgl = TransfList[2] * trgl;
            stk.push(level);
            level.trgl = TransfList[1] * trgl;
            stk.push(level);
            trgl = TransfList[0] * trgl;
        }

        trgl = trgl.t();
        int n_rows = trgl.n_rows;
        GLfloat **poly = toGLfloatPoints(trgl, n_rows);
        drawPolygone(poly, n_rows);

        if (stk.empty())
            break;
        else
        {
            level = stk.top();
            stk.pop();
            trgl = level.trgl;
            iter = level.iteration;
            TransfList = level.tList;
        }
    }
}

void getListTransform(arma::Mat<GLfloat> trgl, vector<arma::Mat<GLfloat>> TransfList, int iter)
{
    stack<transformLevel> stk;
    transformLevel level, l1, l2;
    arma::Mat<GLfloat> t;

    while (true)
    {
        while (iter > 0)
        {
            iter--;
            level.iteration = iter;
            l2 = level;
            l2.trans.push_back(2);
            stk.push(l2);
            l1 = level;
            l1.trans.push_back(1);
            stk.push(l1);
            level.trans.push_back(0);
        }

        t = trgl;
        for (auto i = level.trans.cbegin(); i != level.trans.cend(); ++i)
        {
            t = TransfList[*i] * t;
        }

        t = t.t();
        int n_rows = t.n_rows;
        GLfloat **poly = toGLfloatPoints(t, n_rows);
        drawPolygone(poly, n_rows);

        if (stk.empty())
            break;
        else
        {
            level = stk.top();
            stk.pop();
            iter = level.iteration;
        }
    }
}

void getListTransform2(vector<vector<int>> &Tlist, int iter)
{
    stack<transformLevel> stk;
    transformLevel level, l1, l2;
    arma::Mat<GLfloat> t;

    while (true)
    {
        while (iter > 0)
        {
            iter--;
            level.iteration = iter;
            l2 = level;
            l2.trans.push_back(2);
            stk.push(l2);
            l1 = level;
            l1.trans.push_back(1);
            stk.push(l1);
            level.trans.push_back(0);
        }

        Tlist.push_back(level.trans);

        if (stk.empty())
            break;
        else
        {
            level = stk.top();
            stk.pop();
            iter = level.iteration;
        }
    }
}

void divideTriangleIterative2(arma::Mat<GLfloat> trgl, vector<arma::Mat<GLfloat>> TransfList, int iter)
{
    vector<vector<int>> Tlist;
    arma::Mat<GLfloat> t;
    getListTransform2(Tlist, iter);
    for (int j = 0; j < Tlist.size(); j++)
    {
        t = trgl;
        for (auto i = Tlist[j].cbegin(); i != Tlist[j].cend(); ++i)
        {
            t = TransfList[*i] * t;
        }

        t = t.t();
        int n_rows = t.n_rows;
        GLfloat **poly = toGLfloatPoints(t, n_rows);
        drawPolygone(poly, n_rows);
    }
}

void display()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glScalef(zoom, zoom, zoom);
    random(true);
    //divideTriangle(Triangle2, transfMat, iterations);
    //divideTriangleIterative(Triangle2, transfMat, iterations);
    //getListTransform(Triangle2, transfMat, iterations);
    divideTriangleIterative2(Triangle2, transfMat, iterations);
    glFlush();
}

int main(int argc, char **argv)
{

    generateColors();

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
    glutInitWindowSize(600, 600);
    glutInitWindowPosition(0, 0);
    glutCreateWindow("Sierpinski Triangle");
    glutPositionWindow(100, 100);

    glutKeyboardFunc(keyboard);
    //glutSpecialFunc(special);
    glutMouseFunc(mouse);

    glutCreateMenu(menu);
    glutAddMenuEntry("Smooth shading", MENU_SMOOTH);
    glutAddMenuEntry("Flat shading", MENU_FLAT);
    glutAttachMenu(GLUT_RIGHT_BUTTON);

    init();
    glutDisplayFunc(display);
    glutMainLoop();

    return 0;
}

GLfloat **toGLfloatPoints(arma::Mat<GLfloat> armapoly, int n_row)
{
    GLfloat **poly = (GLfloat **)malloc(n_row * sizeof(GLfloat *));
    for (int i = 0; i < n_row; i++)
    {
        poly[i] = (GLfloat *)malloc(3 * sizeof(GLfloat));
        for (int j = 0; j < 3; j++)
        {
            poly[i][j] = armapoly(i, j);
        }
    }
    return poly;
}

void drawPolygone(GLfloat **poly, int n_row)
{
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    glShadeModel(shading);
    glBegin(GL_TRIANGLES);
    for (int i = 0; i < n_row; i++)
    {
        glColor3f(random(false), random(false), random(false));
        glVertex3fv(poly[i]);
    }
    glEnd();
}

void generateColors()
{
    for (int i = 0; i < 1000; i++)
    {
        colors[i] = rand() / (double)RAND_MAX;
    }
}

double random(bool reset)
{
    static int curr = 0;
    if (reset)
    {
        curr = 0;
        return 0.0;
    }
    else
    {
        if (curr >= 1000)
            curr = 0;
        return colors[curr++];
    }
}

void keyboard(unsigned char key, int x, int y)
{
    switch (key)
    {
    case '+':
        if (iterations < maxIteration)
            iterations += 1;
        display();
        break;
    case '-':
        if (iterations > 0)
            iterations -= 1;
        display();
        break;
    case 'q':
        exit(0);
        break;
    }
}

void mouse(int button, int state, int x, int y)
{
    if ((button == 3) || (button == 4)) // It's a wheel event
    {
        if (button == 3)
        {
            zoom += 0.5;
        }
        else if (button == 4)
        {
            if (zoom >= 1.5)
                zoom -= 0.5;
            else
                zoom = 1;
        }
        display();
    }
    else
    { // normal button event
        //if (button == GLUT_LEFT_BUTTON){

        if (state == GLUT_UP)
        {
            generateColors();
            display();
        }
    }
}

void menu(int item)
{
    switch (item)
    {
    case MENU_FLAT:
        shading = GL_FLAT;
        display();
        break;
    case MENU_SMOOTH:
        shading = GL_SMOOTH;
        display();
        break;
    }
}

void init()
{
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glColor3f(0.0, 0.0, 0.0);
    glMatrixMode(GL_PROJECTION);
    gluPerspective(30, 1, 0.1, 500);
    gluLookAt(2, 2, 2, 0, 0.2, 0, 0, 1, 0);
}
