#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>

#include <hip/hip_runtime.h>

#include <armadillo>

#include <GL/glut.h>

using namespace arma;
using namespace std;

#define N (2042 * 2042)
#define THREADS_PER_BLOCK 512

GLfloat xRotated, yRotated, zRotated;

void init(void);
void DrawCube(void);
void animation(void);
void reshape(int x, int y);

void randomInts(int *a, int n)
{
  int i;
  for (i = 0; i < n; i++)
  {
    a[i] = rand()%(10000-100 + 1) + 100;
  }
}

void saveToFile(FILE *fp, int *a, int *b, int *c)
{
  for (int i = 0; i < 10; i++)
  {
    fprintf(fp, "%d + %d = %d\n", a[i], b[i],c[i]);
  }
}

__global__ 
void add(int *a, int *b, int *c)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

int main(int argc, char **argv)
{
  
  cout << "Armadillo version: " << arma_version::as_string() << endl;
  
  
  int *a, *b, *c;
  int *d_a, *d_b, *d_c; 
  int size = N * sizeof(int);
  
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Setup input values
  a = (int *)malloc(size);
  randomInts(a, N);
  b = (int *)malloc(size);
  randomInts(b, N);
  c = (int *)malloc(size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  add<<<N / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  FILE *fp;

  fp = fopen("result.txt","w");
  saveToFile(fp,a,b,c);
  fclose(fp);

  // Cleanup
  free(a);
  free(b);
  free(c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  glutInit(&argc, argv);
  //we initizlilze the glut. functions
  glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
  glutInitWindowPosition(100, 100);
  glutCreateWindow("glut openGl and cuda");

  //info version GLSL
  cout << "***** Info GPU *****" << std::endl;
  cout << "Fabricant : " << glGetString(GL_VENDOR) << std::endl;
  cout << "Carte graphique: " << glGetString(GL_RENDERER) << std::endl;
  cout << "Version : " << glGetString(GL_VERSION) << std::endl;
  cout << "Version GLSL : " << glGetString(GL_SHADING_LANGUAGE_VERSION) << std::endl;

  init();
  glutDisplayFunc(DrawCube);
  glutReshapeFunc(reshape);
  //Set the function for the animation.
  glutIdleFunc(animation);
  glutMainLoop();
  return 0;
}

void init(void)
{
  glClearColor(0, 0, 0, 0);
  glCullFace(GL_BACK);
  glEnable(GL_CULL_FACE);
}

void DrawCube(void)
{

  glMatrixMode(GL_MODELVIEW);
  // clear the drawing buffer.
  glClear(GL_COLOR_BUFFER_BIT);
  glLoadIdentity();
  glTranslatef(0.0, 0.0, -10.5);
  glRotatef(xRotated, 1.0, 0.0, 0.0);
  // rotation about Y axis
  glRotatef(yRotated, 0.0, 1.0, 0.0);
  // rotation about Z axis
  glRotatef(zRotated, 0.0, 0.0, 1.0);
  glBegin(GL_QUADS);               // Draw The Cube Using quads
  glColor3f(0.0f, 1.0f, 0.0f);     // Color Blue
  glVertex3f(1.0f, 1.0f, -1.0f);   // Top Right Of The Quad (Top)
  glVertex3f(-1.0f, 1.0f, -1.0f);  // Top Left Of The Quad (Top)
  glVertex3f(-1.0f, 1.0f, 1.0f);   // Bottom Left Of The Quad (Top)
  glVertex3f(1.0f, 1.0f, 1.0f);    // Bottom Right Of The Quad (Top)
  glColor3f(1.0f, 0.5f, 0.0f);     // Color Orange
  glVertex3f(1.0f, -1.0f, 1.0f);   // Top Right Of The Quad (Bottom)
  glVertex3f(-1.0f, -1.0f, 1.0f);  // Top Left Of The Quad (Bottom)
  glVertex3f(-1.0f, -1.0f, -1.0f); // Bottom Left Of The Quad (Bottom)
  glVertex3f(1.0f, -1.0f, -1.0f);  // Bottom Right Of The Quad (Bottom)
  glColor3f(1.0f, 0.0f, 0.0f);     // Color Red
  glVertex3f(1.0f, 1.0f, 1.0f);    // Top Right Of The Quad (Front)
  glVertex3f(-1.0f, 1.0f, 1.0f);   // Top Left Of The Quad (Front)
  glVertex3f(-1.0f, -1.0f, 1.0f);  // Bottom Left Of The Quad (Front)
  glVertex3f(1.0f, -1.0f, 1.0f);   // Bottom Right Of The Quad (Front)
  glColor3f(1.0f, 1.0f, 0.0f);     // Color Yellow
  glVertex3f(1.0f, -1.0f, -1.0f);  // Top Right Of The Quad (Back)
  glVertex3f(-1.0f, -1.0f, -1.0f); // Top Left Of The Quad (Back)
  glVertex3f(-1.0f, 1.0f, -1.0f);  // Bottom Left Of The Quad (Back)
  glVertex3f(1.0f, 1.0f, -1.0f);   // Bottom Right Of The Quad (Back)
  glColor3f(0.0f, 0.0f, 1.0f);     // Color Blue
  glVertex3f(-1.0f, 1.0f, 1.0f);   // Top Right Of The Quad (Left)
  glVertex3f(-1.0f, 1.0f, -1.0f);  // Top Left Of The Quad (Left)
  glVertex3f(-1.0f, -1.0f, -1.0f); // Bottom Left Of The Quad (Left)
  glVertex3f(-1.0f, -1.0f, 1.0f);  // Bottom Right Of The Quad (Left)
  glColor3f(1.0f, 0.0f, 1.0f);     // Color Violet
  glVertex3f(1.0f, 1.0f, -1.0f);   // Top Right Of The Quad (Right)
  glVertex3f(1.0f, 1.0f, 1.0f);    // Top Left Of The Quad (Right)
  glVertex3f(1.0f, -1.0f, 1.0f);   // Bottom Left Of The Quad (Right)
  glVertex3f(1.0f, -1.0f, -1.0f);  // Bottom Right Of The Quad (Right)
  glEnd();                         // End Drawing The Cube
  glFlush();
}

void animation(void)
{

  yRotated += 0.01;
  xRotated += 0.02;
  DrawCube();
}

void reshape(int x, int y)
{
  if (y == 0 || x == 0)
    return; //Nothing is visible then, so return
  //Set a new projection matrix
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  //Angle of view:40 degrees
  //Near clipping plane distance: 0.5
  //Far clipping plane distance: 20.0

  gluPerspective(40.0, (GLdouble)x / (GLdouble)y, 0.5, 20.0);
  glMatrixMode(GL_MODELVIEW);
  glViewport(0, 0, x, y); //Use the whole window for rendering
}